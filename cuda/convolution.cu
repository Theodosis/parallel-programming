#include "hip/hip_runtime.h"
#define BLOCKSIZE 64

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
using namespace std;

void cuda_safe( hipError_t error, char* message ){
    if( error != hipSuccess ){
       printf( "Error: %s : %s\n", message, hipGetErrorString( error ) );
    }
}
void calcTime( struct timeval first, struct timeval second, struct timeval *lapsed ){
    if(first.tv_usec>second.tv_usec){
        second.tv_usec += 1000000;
        second.tv_sec--;
    }
    lapsed->tv_usec = second.tv_usec - first.tv_usec;
    lapsed->tv_sec = second.tv_sec - first.tv_sec;
}
void randomize( float *a, int x, int y ){
    int i, j = 0;
    srand(time(NULL)); // generate different random numbers
    for(i=0; i<x; i++){
        for(j=0; j<y; j++){
            a[ i * y + j ] = (float) rand() / RAND_MAX;
        }
    }
}

void zeroes( float *a, int x, int y ){
    int i, j;
    for( i = 0; i < x; ++i ){
        for( j = 0; j < y; ++j ){
            a[ i * y + j ] = 0;
        }
    }
}

void pad( float *a, float *b, int x, int y, int px, int py ){
    int i, j;
    zeroes( b, x + px, y + py );
    for( i = 0; i < x; ++i ){
        for( j = 0; j < y; ++j ){
            b[ ( i + px ) * ( y + 2 * py ) + ( j + py ) ] = a[ i * y + j ];
        }
    }
}

void print( float* a, int x, int y ){
    int i, j;
    for( i = 0; i < x; ++i ){
        for( j = 0; j < y; ++j ){
            printf( "%f", a[ i * y + j ] );
            printf( j == y - 1 ? "" : ", " );
        }
        printf( ";\n" );
    }
    printf( "\n" );
}

void sconv( float* pF, float* T, float* Y, int mpF, int npF, int mT, int nT, int mY, int nY ){
    int i, j, k, l, plF, plTF, plT;
    for( i = 0; i < mY; ++i ){
        for( j = 0; j < nY; ++j ){
            for( k = 0; k < mT; ++k ){
                for( l = 0; l < nT; ++l ){
                    plF = i * npF + j;
                    plTF = k * npF + l;
                    plT = k * nT + l;
                    Y[ i * nY + j ] += pF[ plF + plTF ] * T[ plT ];
                }
            }
        }
    }
}

__global__ void cconv( float *cpF, float *cT, float *cY, int mpF, int npF, int mT, int nT, int mY, int nY ){
    int k, l, plF;
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int i = ( int ) ix / nY;
    int j = ix - i * nY;
    plF = i * npF + j;
    float sum = 0; 

    __shared__ float sT[ 24 * 24 ];
    
    for( k = 0; k < 10; ++k ){
        if( threadIdx.x * 10 + k >= mT * nT ){
            break;
        }
        sT[ threadIdx.x * 10 + k ] = cT[ threadIdx.x * 10 + k ];
    }
    __syncthreads();
    
    for( k = 0; k < mT; ++k ){
        for( l = 0; l < nT; ++l ){
            sum += cpF[ plF + k * npF + l ] * sT[ k * nT + l ];
        }
    }
    cY[ i * nY + j ] = sum;
}


int main( int argc, char **argv ){
    if( argc != 5 ){
        printf( "we need 4 int values\n" );
        return 0;
    }
    struct timeval first, second, lapsed, third, forth;
    struct timezone tzp;
    int mF = atoi( argv[ 1 ] ),
        nF = atoi( argv[ 2 ] ),
        mT = atoi( argv[ 3 ] ),
        nT = atoi( argv[ 4 ] ),
        m = mT - 1,
        n = nT - 1,
        mY = mF + m,
        nY = nF + n,
        mpF = mF + 2 * m,
        npF = nF + 2 * n,
        totalsize, blocks,
        mres = 0, mdiff = 0;
    
    totalsize = mY * nY;
    
    blocks = totalsize / BLOCKSIZE;
    
    if( BLOCKSIZE * blocks != totalsize  ){
        mres = mY / BLOCKSIZE;
        mdiff = mY - BLOCKSIZE * mres;
        
        mpF += BLOCKSIZE - mdiff;
        mY += BLOCKSIZE - mdiff;
        totalsize = mY * nY;
        blocks++;// = totalsize / BLOCKSIZE;
    }
    
    float *F,  *T,  *Y, *pF,
          *cpF, *cT, *cY;
    
    F = (float*) malloc( sizeof( float ) * mF * nF  );
    pF = (float*) malloc( sizeof( float ) * mpF * npF );
    T = (float*) malloc( sizeof( float ) * mT * nT );
    Y = (float*) malloc( sizeof( float ) * mY * nY );
    
    cuda_safe( hipMalloc( &cpF, sizeof( float ) * mpF * npF ), "cudaMalloc1" );
    cuda_safe( hipMalloc( &cT, sizeof( float ) * mT * nT ), "cudaMalloc2" );
    cuda_safe( hipMalloc( &cY, sizeof( float ) * mY * nY ), "cudaMalloc3" );
    
    randomize( F, mF, nF );
    randomize( T, mT, nT );
    zeroes( Y, mY, nY );
    pad( F, pF, mF, nF, m, n );
    cuda_safe( hipMemcpy( cpF, pF, sizeof( float ) * mpF * npF, hipMemcpyHostToDevice ), "cudaMemcpy1" );
    cuda_safe( hipMemcpy( cT, T, sizeof( float ) * mT * nT, hipMemcpyHostToDevice ), "cudaMemcpy2" );
    //cuda_safe( hipMemcpy( cY, Y, sizeof( float ) * mY * nY, hipMemcpyHostToDevice ), "cudaMemcpy3" );

    gettimeofday(&first, &tzp);
    sconv( pF, T, Y, mpF, npF, mT, nT, mY, nY );
    gettimeofday(&second, &tzp);
    
    gettimeofday(&third, &tzp);
    cconv<<< blocks, BLOCKSIZE >>> ( cpF, cT, cY, mpF, npF, mT, nT, mY, nY );
    hipDeviceSynchronize();
    gettimeofday(&forth, &tzp);
    
    cuda_safe( hipMemcpy( Y, cY, sizeof( float ) * mY * nY, hipMemcpyDeviceToHost ), "hipMemcpy" );
    
    //print( F, mF, nF );
    //print( T, mT, nT );
    //print( Y, mY, nY );
    

    calcTime( first, second, &lapsed );
    printf("%d.%06d ", (int) lapsed.tv_sec, (int) lapsed.tv_usec);  

    calcTime( third, forth, &lapsed );
    printf("%d.%06d\n", (int) lapsed.tv_sec, (int) lapsed.tv_usec);  
    
    
    //cuda_safe( hipFree( cpF ), "hipFree" );
    //cuda_safe( hipFree( cT ), "hipFree" );
    //cuda_safe( hipFree( cY ), "hipFree" );
    free( F );
    free( pF );
    free( T );
    free( Y );
    return 0;
}

